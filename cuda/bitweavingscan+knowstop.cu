#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <string.h>
#include <unistd.h>
#include <time.h>
#include <algorithm>
#include <vector>
#include "scanImpl.cu"
#include "../include/common.h"
#include "../include/gpuCudaLib.h"

using namespace std;

#define CHECK_POINTER(p)   do {                     \
    if(p == NULL){                                  \
        perror("Failed to allocate host memory");   \
        exit(-1);                                   \
    }} while(0)
const int know_stop_size=1000010;
vector<int > know_stop_num[know_stop_size];
int nlz(unsigned x){
   int n;

   if (x == 0) return(32);
   n = 1;
   if ((x >> 16) == 0) {n = n +16; x = x <<16;}
   if ((x >> 24) == 0) {n = n + 8; x = x << 8;}
   if ((x >> 28) == 0) {n = n + 4; x = x << 4;}
   if ((x >> 30) == 0) {n = n + 2; x = x << 2;}
   n = n - (x >> 31);
   return n;
}
__global__ void static equal(int * a, int n, unsigned int constC){
    int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i=offset; i<n; i+=stride){    
        a[i] = constC;
    }
}


__global__ void static genScanFilter_int_lth_bit(int * col,int n, unsigned int constC,int * lt, int * eq){
    int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  

    for(int i=offset; i < n; i+=stride){
        lt[i] = lt[i] | (eq[i] & ~constC & col[i]);
        eq[i] = eq[i] & ~(col[i] ^ constC);
        //printf(" %d %u %u %u\n",i,lt[i],eq[i],col[i]);
    }
}
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}
int bit_constC(int where,int j){

        int constC = 0;
        for(int k = 0;k < 32;++k) {
            constC += ((((1U << (31 - j )) & where)>>(31-j))<< k);
          
        }
        return constC;
}
void profilebitweavscan(int        *h_a, 
                   int        *h_b, 
                   int        *d, 
                   int *lt,
                   int *eq,
                   int *know_stop_len_cpu,
                   int *know_stop_constC_cpu,
                   int early_size,
                   int  n,
                   char         *desc,
                   unsigned int loopTotal)
{

  dim3 block(256);
  dim3 grid(2048);
  float time,stime;
  // events for timing
  hipEvent_t startEvent, stopEvent; 
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) ); 
  stime=0;

  for(int loop = 1; loop <= loopTotal; loop++){  
      checkCuda( hipEventRecord(startEvent, 0) );

      unsigned int c = 0;
      for(int i = 0;i < 32;i++) c += (1u << i);
      equal<<<grid,block>>>(lt, n/32, 0) ;
      equal<<<grid,block>>>(eq, n/32, c) ;
      checkCuda(hipDeviceSynchronize());
      // printf("%d\n",    clock());
      for(int j = 0; j < 32; ++j){

            for(int k  = 0; k < n / 32; k += early_size / 32){

                //printf("%d \n", know_stop_len_cpu[k / (early_size / 32)]);                
                if( j > know_stop_len_cpu[k / (early_size / 32)]) continue;

                unsigned int constC = bit_constC(know_stop_constC_cpu[k / (early_size / 32)], j); 
                checkCuda( hipMemcpy(d + j * (n / 32) + k, h_a + j * (n / 32) + k, early_size / 32 *4, hipMemcpyHostToDevice) );
                genScanFilter_int_lth_bit<<<grid,block>>>(d + j * (n / 32) + k, early_size / 32,  constC, lt + k, eq + k);
                checkCuda(hipDeviceSynchronize());
            }


      }

      checkCuda( hipMemcpy(h_b, lt, n / 32 * 4, hipMemcpyDeviceToHost) );
      checkCuda( hipMemcpy(h_b + n / 32 , eq, n / 32 * 4, hipMemcpyDeviceToHost) );
      checkCuda( hipEventRecord(stopEvent, 0) );
      checkCuda( hipEventSynchronize(stopEvent) );

      checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
      stime += time;
      //printf("%f\n",stime);
  }
  int bytes=n * sizeof(int);
  printf("%f\n" ,bytes * 1e-6/(stime / loopTotal));
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
}
int main(int argc, char ** argv)
{
  dim3 block(256);
  dim3 grid(2048);
  int inputN;
  sscanf(argv[1],"%d",&inputN);
  unsigned int nElements = inputN;
  const unsigned int bytes = nElements * sizeof(int);

  // host arrays
  int *h_aPageable, *h_bPageable,*h_bitPageable,*know_stop_len_cpu,*know_stop_constC_cpu;   
  int *h_aPinned, *h_bPinned;

  // device array
  int *d_a,*lt,*eq;

  // allocate and initialize
  h_aPageable = (int*)malloc(bytes );          
  h_bPageable = (int*)malloc(bytes );
  h_bitPageable =(int *)malloc(bytes );
  know_stop_len_cpu = (int *)malloc(bytes );     
  know_stop_constC_cpu = (int *)malloc(bytes );           // host pageable
  checkCuda( hipHostMalloc((void**)&h_aPinned, bytes  ) ); // host pinned
  checkCuda( hipHostMalloc((void**)&h_bPinned, bytes  ) );  
  checkCuda( hipMalloc((void**)&d_a, bytes  ) );           // device
  checkCuda( hipMalloc((void**)&lt, bytes ) ); // device return
  checkCuda( hipMalloc((void**)&eq, bytes  ) );  

  int early_size = 1024*1024; 
  sscanf(argv[2],"%d",&early_size);  
  srand(0);
  for (int i = 0; i < nElements; ++i) h_aPageable[i] = rand()%(1U<<5);  

  for   (int i = 0;i < nElements/ early_size; i++){
        know_stop_num[i].push_back(0);
        know_stop_num[i].push_back((1U<<31)-1);
        //0----2^31-1
        for(int j=0;j < early_size; j++)
          know_stop_num[i].push_back(h_aPageable[i * early_size + j]);
        sort(know_stop_num[i].begin(),know_stop_num[i].end());
        know_stop_num[i].erase(unique(know_stop_num[i].begin(), know_stop_num[i].end()), know_stop_num[i].end());
  }   
     for(int j = 31; j >= 0; --j){
        for (int i = 0; i < nElements; ++i) 

          h_bitPageable[i / 32 + (31-j)*(nElements/32)] += (((h_aPageable[i] &(1<<j))>>j)<<(31 - i % 32));
     }
  
  memcpy(h_aPinned, h_aPageable, bytes  );
  memset(h_bPageable, 0, bytes);
  memset(h_bPinned, 0, bytes);
  memset(know_stop_len_cpu, 0, bytes);

  // output device info and transfer size
  hipDeviceProp_t prop;

  checkCuda( hipGetDeviceProperties(&prop, 0) );



  int constC = rand()%(1U<<5);
  for(int i = 0 ; i < nElements/early_size; ++i){
        assert(constC > 0);
        know_stop_constC_cpu[i] = *lower_bound(know_stop_num[i]. begin(),know_stop_num[i].end(),constC);
        int last_constC = *(--lower_bound(know_stop_num[i]. begin(),know_stop_num[i].end(),constC));

        know_stop_len_cpu[i] =nlz(last_constC ^ know_stop_constC_cpu[i]) + 1; 
       // printf("haha=%d %d %d\n",know_stop_len_cpu[i] ,last_constC,know_stop_constC_cpu[i], know_stop_len_cpu[i]);
  }


  // perform  scan eq
 // profilescan(h_aPageable, h_bPageable, d_a, filter, nElements, constC,"Pageable",20);
  //profilescan(h_aPinned, h_bPinned, d_a, filter,nElements, constC,"Pinned",20);

  profilebitweavscan(h_bitPageable, h_bPageable, d_a, lt, eq, know_stop_len_cpu, know_stop_constC_cpu,early_size, nElements,"Pageable",1);
  // printf("constC=%d\n",constC);
  // for(int i = 0; i < nElements; i++) printf("%u ",h_aPageable[i]);printf("\n");
  // for(int i = 0; i < nElements; i++) printf("%u ",((h_bPageable[i/32] & (1u << (31 - i % 32)))>> (31 - i % 32)));printf("\n");
  // for(int i = 0; i < nElements; i++) printf("%u ",((h_bPageable[i/32 + nElements/32] & (1u << (31 - i % 32)))>> (31 - i % 32)));printf("\n");
  // //for(int i = 0; i < nElements; i++) printf("%3u ",h_bitPageable[i]);printf("\n");

  // cleanup
  hipFree(lt);
  hipFree(eq);
  hipHostFree(h_aPinned);
  hipHostFree(h_bPinned);
  free(h_aPageable); 
}
