#ifndef SCAN_IMPL_CU
#define SCAN_IMPL_CU

#include "scan.cu"
#include "../include/common.h"
#include "../include/gpuCudaLib.h"

static void scanImpl(int *d_input, int rLen, int *d_output, struct statistic * pp)
{
    int len = 2;
    if(rLen < len){
            int *input, *output;
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&input,len*sizeof(int)));
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&output, len*sizeof(int)));
            CUDA_SAFE_CALL_NO_SYNC(hipMemset(input, 0, len*sizeof(int)));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(input, d_input, rLen*sizeof(int), hipMemcpyDeviceToDevice));
            preallocBlockSums(len);
            prescanArray(output, input, len, pp);
            deallocBlockSums();
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_output,output,rLen*sizeof(int),hipMemcpyDeviceToDevice));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(input));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(output));
            return;
    }else{
            preallocBlockSums(rLen);
            prescanArray(d_output, d_input, rLen, pp);
            deallocBlockSums();
    }
//	preallocBlockSums(rLen);
//	prescanArray(d_output, d_input, rLen, pp);
//	deallocBlockSums();
}


#endif

